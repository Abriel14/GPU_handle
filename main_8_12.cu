#include "hip/hip_runtime.h"
#include <iostream>
#include <bit>
#include <bitset>
#include "Data_8_12_0.cpp"

#define NBR_RIDGES 760 //first multiple of 76 larger than 708
#define NBR_LOOPS 1 //out of 121
#define RESULT_SIZE (1u<<13)
#define BLOCK_SIZE 152
#define SUB_BLOCK 2
#define DIVISOR 16
#define CUDA_CORES 3584
using namespace std;

struct StructX0 {
    unsigned long X0 = 0ul;
    unsigned int precalc[10] = {0u};
};
const int nbrX0 = NBR_X0;
const int nbrX1 = NBR_X1;
__shared__ int r[NBR_RIDGES];
unsigned int ai_host[SUB_BLOCK][BLOCK_SIZE];
__device__ unsigned int ai_device[SUB_BLOCK][BLOCK_SIZE];
int mi_host[SUB_BLOCK][N][BLOCK_SIZE];
__device__ int mi_device[SUB_BLOCK][N][BLOCK_SIZE];
unsigned int X1_host[nbrX1];
__device__ unsigned int X1_device[nbrX1];
unsigned long out_host[CUDA_CORES][RESULT_SIZE];
__device__ __managed__ unsigned long out_device[CUDA_CORES][RESULT_SIZE];
int n_out_host[CUDA_CORES]={};
__device__ __managed__ int n_out_device[CUDA_CORES]={};
StructX0 host_listX0[nbrX0];
__device__ __managed__ StructX0 device_listX0[nbrX0];

__global__ void kernel(StructX0 structX0[]) {
    unsigned int a[SUB_BLOCK];
    unsigned int precalc_a = structX0[blockIdx.x].precalc[threadIdx.x / DIVISOR];
    unsigned long X0 = structX0[blockIdx.x].X0;
    for (int k = 0; k < SUB_BLOCK; k++) {
        a[k] = ai_device[k][threadIdx.x] | ((precalc_a >> (SUB_BLOCK * (threadIdx.x % DIVISOR) + k)) & 1u << 31);
    }
    int m[SUB_BLOCK][N];
    for (int k = 0; k < SUB_BLOCK; k++) {
        for (int l = 0; l < N; l++) {
            m[k][l] = mi_device[k][l][threadIdx.x];
        }
    }
    int count;
    bool Ax[SUB_BLOCK];
    bool stop;
    for (unsigned int X1: X1_device) {
        stop = false;
        for (int i = 0; i < NBR_RIDGES; i += BLOCK_SIZE) r[i + threadIdx.x] = 0;
        __syncthreads();
        for (int j = 0; j < SUB_BLOCK; j++) {
            Ax[j] = __popc(a[j] & X1) & 1;
        }
        count = 0;
        for (bool j: Ax) {
            count += __syncthreads_count(j);
        }
        if (count > MAX_NBR_FACETS) continue;
        for (int k = 0; k < SUB_BLOCK; k++) {
            if (stop) break;
            if (Ax[k]) {
                for (int t = 0; t < N; t++) {
                    if (atomicAdd(r+m[k][t], 1) >= 2) {
                        stop = true;
                        break;
                    }
                }
            }
        }
        if (__syncthreads_or(stop)) continue;
        if (threadIdx.x == 0) {
            out_device[blockIdx.x%CUDA_CORES][atomicAdd(n_out_device+(blockIdx.x%CUDA_CORES), 1)] = (X0 | (unsigned long) (X1 ^ (1u << 31)));
        }
    }

}

void increment_vect(unsigned int vect[], const unsigned int ref[], const int starting_index, const int size) {
    vect[0] = (vect[0] + 1) % ref[starting_index];
    int k = 0;
    while (vect[k] == 0 and k < size - 1) {
        k += 1;
        vect[k] = (vect[k] + 1) % ref[starting_index + k];
    }
}


int main() {
    unsigned int vectX0[sizeVectX0];
    for (int k = 0; k < sizeVectX0; k++) vectX0[k] = 0;
    unsigned int vectX1[sizeVectX1];
    for (int k = 0; k < sizeVectX1; k++) vectX1[k] = 0;
    unsigned int list_shifts[NBR_GROUPS];
    unsigned int list_ref[NBR_GROUPS - 1];
    unsigned long list_elementary[NBR_GROUPS - 1][11] = {};
    unsigned long X0;

    //Initialiser les matrices ai et mi
    for (int k = 0; k < NBR_FACETS; k++) {
        ai_host[k % SUB_BLOCK][k / SUB_BLOCK] = (unsigned int) ((A[k] << 33) >> 33);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(&ai_device), &ai_host, sizeof(ai_host));
    for (int k = 0; k < NBR_FACETS; k++) {
        for (int l = 0; l < N; l++) {
            mi_host[k % SUB_BLOCK][l][k / SUB_BLOCK] = M[l][k];
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(&mi_device), &mi_host, sizeof(mi_host));
    //Initialiser les shifts et les générateurs de combinaison linéaire
    list_shifts[NBR_GROUPS - 1] = 64 - NBR_GENERATORS;
    for (int k = NBR_GROUPS - 2; k > -1; k--) {
        list_shifts[k] = list_groups[k + 1] + list_shifts[k + 1];
    }
    for (int i = 1; i < NBR_GROUPS; i++) {
        int position = 0;
        for (int j = 0; j < (1ul << (list_groups[i])); j++) {
            if (__popcount(j) <= 2) {
                unsigned long jl = j;
                list_elementary[i - 1][position] = (jl << list_shifts[i]);
                position += 1;
            }
        }
        list_ref[i - 1] = position;
    }
    //Initialiser les X1
    unsigned int X1;
    for (unsigned int &X1_val: X1_host) {
        X1 = 1u << 31;
        for (int i = 0; i < sizeVectX1; i++) {
            X1 |= (unsigned int) (list_elementary[i + sizeVectX0][vectX1[i]]);
        }
        X1_val = X1;
        increment_vect(vectX1, list_ref, sizeVectX0, sizeVectX1);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(&X1_device), &X1_host, sizeof(X1_host));
    bool last_one_copied[CUDA_CORES];
    bool first_appeared;
    for (int l = 0; l < NBR_LOOPS; l++) {
        for(bool x: last_one_copied) x=false;
        for (int index = 0; index < nbrX0; index++) {
            X0 = (1ul << 63);
            for (int i = 0; i < sizeVectX0; i++) {
                X0 |= list_elementary[i][vectX0[i]];
            }
            host_listX0[index].X0 = X0;
            increment_vect(vectX0, list_ref, 0, sizeVectX0);
        }
        for (int index = 0; index < nbrX0; index++) {
            for (int i = 0; i < BLOCK_SIZE; i++) {
                for (int k = 0; k < SUB_BLOCK; k++) {
                    if ((__popcount(host_listX0[index].X0 & A[i * SUB_BLOCK + k])) & 1u)
                        host_listX0[index].precalc[i / DIVISOR] |= 1u << (SUB_BLOCK * (i % DIVISOR) + k);
                }
            }
        }
        hipMemcpyToSymbol(HIP_SYMBOL(&device_listX0), &host_listX0, sizeof(host_listX0));
        kernel<<<NBR_X0, BLOCK_SIZE>>>(device_listX0);
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
            printf("kernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
        hipMemcpyFromSymbol(&n_out_host, HIP_SYMBOL(&n_out_device), sizeof(n_out_device));
        for (unsigned int n:n_out_host){
            if (n_out_host[n] > (1u << 12)) {
                hipMemcpyFromSymbol(&out_host[n], HIP_SYMBOL(&out_device[n]), sizeof(out_device[n]));
                for (int i = 0; i < n_out_host[n]; i++) {
                    first_appeared = false;
                    cout<<'[';
                    for (int j=0;j<NBR_FACETS;j++){
                        if (__popcount(out_host[n][i]&A[j])&1ul){
                            if(first_appeared) cout<<',';
                            first_appeared = true;
                            cout<<F[j];
                        }
                    }
                    cout<<']'<<'\n';
                }
                n_out_host[n] = 0;
                last_one_copied[n]=true;
                hipMemcpyToSymbol(HIP_SYMBOL(&n_out_device[n]), &n_out_host[n], sizeof(n_out_host[n]));
            }
        }

    }
    hipMemcpyFromSymbol(&n_out_host, HIP_SYMBOL(&n_out_device), sizeof(n_out_device));
    hipMemcpyFromSymbol(&out_host, HIP_SYMBOL(&out_device), sizeof(out_device));
    for (unsigned int n:n_out_host) {
        if (not last_one_copied[n]) {
            for (int i = 0; i < n_out_host[n]; i++) {
                cout << '[';
                first_appeared = false;
                for (int j = 0; j < NBR_FACETS; j++) {
                    if (__popcount(out_host[n][i] & A[j]) & 1ul) {
                        if (first_appeared) cout << ',';
                        first_appeared = true;
                        cout << F[j];
                    }
                }
                cout << ']' << '\n';
            }
        }
    }
    return 0;
}