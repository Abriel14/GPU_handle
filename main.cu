
#include <hip/hip_runtime.h>
#include <iostream>
#include <bit>
#include <bitset>
#include <cstdint>

#define MAXFACES 252
#define SIZE_X 198414832
#define RESULT_SIZE (1ul<<32)

using namespace std;

__shared__ int r[2688];
__device__  unsigned int ai[4][210];
__device__ int mi[4][11][210];
__device__ unsigned int X[SIZE_X];
__device__ __managed__ long out[RESULT_SIZE];
__device__ __managed__ int n_out = 0;
struct StructX0 {
    unsigned long X0;
//    unsigned int precalc[1];
};

const unsigned long nbrX0 = 214358881;
unsigned long listX0[nbrX0];


//__global__ void kernel(StructX0 structX0[]) {
//    unsigned int a[4];
//    unsigned int precalc_a = structX0[blockIdx.x].precalc[threadIdx.x / 8];
//    unsigned long X0 = structX0[blockIdx.x].X0;
//    for (int k = 0; k < 4; k++) {
//        a[k] = ai[k][threadIdx.x] | ((precalc_a >> (4 * (threadIdx.x % 8) + k)) & 1u) << 31;
//    }
//    int m[4][11];
//    for (int k = 0; k < 4; k++) {
//        for (int l = 0; l < 11; l++) {
//            m[k][l] = mi[k][l][threadIdx.x];
//        }
//    }
//    bool Ax[4];
//    bool stop;
//    for (unsigned int x: X) {
//        for (int j = 0; j < 4; j++) {
//            Ax[j] = __popc(a[j] & x) & 1;
//        }
//        int count = 0;
//        for (bool j: Ax) {
//            count += __syncthreads_count(j);
//        }
//        if (count > MAXFACES) continue;
//        for (bool j: Ax) {
//            if (j) {
//                for (int k = 0; k < 11; k++) {
//                    if (atomicAdd(r + m[j][k], 1) >= 4) {
//                        stop = true;
//                    }
//                }
//            }
//        }
//        if (__syncthreads_or(stop)) continue;
//        if (threadIdx.x == 0) {
//            out[atomicAdd(&n_out, 1)] = X0 | x;
//        }
//    }
//
//}

//void calculs_GPU(unsigned long A[], unsigned int M[]) {
//    // Enumérer les X0 33
//    unsigned int list_groups[20] = {1, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 1, 1, 1, 1};
//    unsigned int list_shifts[20];
//    for (int k = 18; k > -1; k--) {
//        list_shifts[k] += list_groups[k];
//    }
//    cout<<(list_shifts);
//    unsigned long nbrX0 = 36388725375613;
//    StructX0 listX0[nbrX0];
//    unsigned long k = 0;
//    // Précalculer les 33 precalc sur CPU
//    StructX0 precalcTable[nbrX0];
//    // Enumérer les X1 31
//    // lancer le 210 000 000 précalc correspondant à X0 blocks
//    kernel<<<1, 210>>>(precalcTable);
//    for (int i = 0; i < n_out; i++) {
//        cout << out[i];
//        // écrire dans le fichier texte out[k] (printf)
//    }
//}

void increment_vect(unsigned int vect[], const unsigned int ref[], const int size) {
    vect[0] = (vect[0] + 1) % ref[0];
    int k = 0;
    while (vect[k] == 0 and k < size - 1) {
        k += 1;
        vect[k] = (vect[k] + 1) % ref[k];
    }
}

int main() {
    unsigned int list_groups[20] = {1, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 1, 1, 1, 1};
    unsigned int list_shifts[20];
    unsigned int list_ref[19];
    list_shifts[19] = 0;
    for (int k = 18; k > -1; k--) {
        list_shifts[k] = list_groups[k + 1] + list_shifts[k + 1];
    }
    unsigned long list_elementary[19][11];
    for (int i = 0; i < 19; i++) {
        int position = 0;
        for (int j = 0; j < (1ul << (list_groups[i + 1])); j++) {
            if (__popcount(j) <= 2) {
                unsigned long jl = j;
                list_elementary[i][position] = (jl << list_shifts[i + 1]);
                position += 1;
            }
        }
        list_ref[i] = position;
    }
    int size = 8;
    unsigned int vect[size];
    for (int k = 0; k < size; k++) {
        vect[k] = 0;
    }
    unsigned long k = 0;
    while (k < nbrX0) {
        unsigned long x = (1ul << 63);
        for (int i = 0; i < size; i++) {
            x |= list_elementary[i][vect[i]];
        }
        listX0[k] = x;
        increment_vect(vect, list_ref, size);
        k++;
    }
    for (unsigned int l : vect){
        cout<<l<<',';
    }
}