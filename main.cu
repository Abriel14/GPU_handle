#include "hip/hip_runtime.h"
#include <iostream>
#include <bit>
#include <bitset>
#include <cstdint>
#include "Data.cpp"

#define MAX_NBR_FACETS 252
#define NBR_RIDGES 2688
#define NBR_FACETS 840
#define NBR_X0 1771561
#define NBR_X1 198414832
#define NBR_LOOPS 121
#define RESULT_SIZE (1ul<<20)

using namespace std;

struct StructX0 {
    unsigned long X0 = 0ul;
    unsigned int precalc[27] = {0u};
};

const int nbrX0 = NBR_X0;
const int nbrX1 = NBR_X1;
__shared__ int r[NBR_RIDGES];
unsigned int ai_host[4][210];
__device__ unsigned int ai_device[4][210];
int mi_host[4][11][210];
__device__ int mi_device[4][11][210];
unsigned int X1_host[nbrX1];
__device__ unsigned int X1_device[nbrX1];
unsigned long out_host[RESULT_SIZE];
__device__ __managed__ unsigned long out_device[RESULT_SIZE];
int n_out_host = 0;
__device__ __managed__ int n_out_device = 0;
StructX0 host_listX0[nbrX0];
__device__ __managed__ StructX0 device_listX0[nbrX0];


__global__ void kernel(StructX0 structX0[]) {
    unsigned int a[4];
    unsigned int precalc_a = structX0[blockIdx.x].precalc[threadIdx.x / 8];
    unsigned long X0 = structX0[blockIdx.x].X0;
    for (int k = 0; k < 4; k++) {
        a[k] = ai_device[k][threadIdx.x] | ((precalc_a >> (4 * (threadIdx.x % 8) + k)) & 1u) << 31;
    }
    int m[4][11];
    for (int k = 0; k < 4; k++) {
        for (int l = 0; l < 11; l++) {
            m[k][l] = mi_device[k][l][threadIdx.x];
        }
    }
    bool Ax[4];
    bool stop=false;
    for (unsigned int X1: X1_device) {
        if (threadIdx.x==0) {
            memset(r,0,sizeof(r));
        }
        __syncthreads();
        for (int j = 0; j < 4; j++) {
            Ax[j] = __popc(a[j] & X1) & 1;
        }
        int count = 0;
        for (bool j: Ax) {
            count += __syncthreads_count(j);
        }
        if (count > MAX_NBR_FACETS) continue;
        for (int j=0;j<4;j++) {
            if (stop) continue;
            if (Ax[j]) {
                for (int k = 0; k < 11; k++) {
                    if (atomicAdd(r + m[j][k], 1) >= 4) {
                        stop = true;
                        continue;
                    }
                }
            }
        }
        if (__syncthreads_or(stop)) continue;
        if (threadIdx.x == 0) {
            out_device[atomicAdd(&n_out_device, 1)] = (X0 | (unsigned long) (X1^(1u<<31)));
        }
    }

}

void increment_vect(unsigned int vect[], const unsigned int ref[], const int starting_index, const int size) {
    vect[0] = (vect[0] + 1) % ref[starting_index];
    int k = 0;
    while (vect[k] == 0 and k < size - 1) {
        k += 1;
        vect[k] = (vect[k] + 1) % ref[starting_index + k];
    }
}


int main() {
    int sizeVectX0 = 8;
    unsigned int vectX0[sizeVectX0];
    for (int k = 0; k < sizeVectX0; k++) vectX0[k] = 0;
    int sizeVectX1 = 11;
    unsigned int vectX1[sizeVectX1];
    for (int k = 0; k < sizeVectX1; k++) vectX1[k] = 0;
    unsigned int list_groups[20] = {1, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 1, 1, 1, 1};
    unsigned int list_shifts[20];
    unsigned int list_ref[19];
    unsigned long list_elementary[19][11] = {};
    unsigned long X0;

    //Initialiser les matrices ai et mi
    for (int k = 0; k < NBR_FACETS; k++) {
        ai_host[k % 4][k / 4] = (unsigned int) ((A[k] << 32) >> 32);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(ai_device), ai_host, sizeof(ai_host));
    for (int k = 0; k < NBR_FACETS; k++) {
        for (int l = 0; l < 11; l++) {
            mi_host[k % 4][l][k / 4] = M[l][k];
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(mi_device), mi_host, sizeof(mi_host));
    //Initialiser les shifts et les générateurs de combinaison linéaire
    list_shifts[19] = 0;
    for (int k = 18; k > -1; k--) {
        list_shifts[k] = list_groups[k + 1] + list_shifts[k + 1];
    }
    for (int i = 0; i < 19; i++) {
        int position = 0;
        for (int j = 0; j < (1ul << (list_groups[i + 1])); j++) {
            if (__popcount(j) <= 2) {
                unsigned long jl = j;
                list_elementary[i][position] = (jl << list_shifts[i + 1]);
                position += 1;
            }
        }
        list_ref[i] = position;
    }
    //Initialiser les X1
    unsigned int X1;
    for (unsigned int &X1_val: X1_host) {
        X1 = 1u<<31;
        for (int i = 0; i < sizeVectX1; i++) {
            X1 |= (unsigned int) (list_elementary[i + sizeVectX0][vectX1[i]]);
        }
        X1_val = X1;
        increment_vect(vectX1, list_ref, sizeVectX0, sizeVectX1);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(X1_device), X1_host, sizeof(X1_host));
    for (int l=0;l<NBR_LOOPS;l++){
        for (int index = 0; index < nbrX0; index++) {
            X0 = (1ul << 63);
            for (int i = 0; i < sizeVectX0; i++) {
                X0 |= list_elementary[i][vectX0[i]];
            }
            increment_vect(vectX0, list_ref, 0, sizeVectX0);
        }
        for (int index = 0; index < nbrX0; index++) {
            for (int i=0;i<210;i++){
                for(int k=0;k<4;k++){
                    host_listX0[index].precalc[i/8] |= ((__popcount(host_listX0[index].X0 & A[i*4+k])) & 1u) << (4*(i % 8)+k);
                }
            }
        }
        hipMemcpyToSymbol(HIP_SYMBOL(device_listX0), host_listX0, sizeof(host_listX0));
        kernel<<<1, 210>>>(device_listX0);
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
            printf("kernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
        for(int s=0;s<sizeVectX0;s++){
            cout<<vectX0[s]<<',';
        }
        cout<<'\n';
    }
    hipMemcpyFromSymbol(&n_out_host, HIP_SYMBOL(n_out_device), sizeof(n_out_device));
    hipMemcpyFromSymbol(&out_host, HIP_SYMBOL(out_device), sizeof(out_device));
    cout<<(n_out_host)<<'\n';
    for (int i = 0; i < n_out_host; i++) {
        cout << out_host[i]<<'\n';
    }
}