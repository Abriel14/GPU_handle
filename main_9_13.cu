#include "hip/hip_runtime.h"
#include <iostream>
#include <bit>
#include <bitset>
#include <cstdint>
#include "Data_9_13_6.cpp"

#define NBR_RIDGES 1210 //first multiple of 110 larger than 1190
#define NBR_LOOPS 121 //out of 1
#define RESULT_SIZE (1u<<25)
#define BLOCK_SIZE 110

using namespace std;

struct StructX0 {
    unsigned long X0 = 0ul;
    unsigned int precalc[14] = {0u};
};
const int nbrX0 = NBR_X0;
const int nbrX1 = NBR_X1;
__shared__ int r[NBR_RIDGES];
unsigned int ai_host[4][BLOCK_SIZE];
__device__ unsigned int ai_device[4][BLOCK_SIZE];
int mi_host[4][N][BLOCK_SIZE];
__device__ int mi_device[4][N][BLOCK_SIZE];
unsigned int X1_host[nbrX1];
__device__ unsigned int X1_device[nbrX1];
unsigned long out_host[RESULT_SIZE];
__device__ __managed__ unsigned long out_device[RESULT_SIZE];
int n_out_host = 0;
__device__ __managed__ int n_out_device = 0;
StructX0 host_listX0[nbrX0];
__device__ __managed__ StructX0 device_listX0[nbrX0];


__global__ void kernel(StructX0 structX0[]) {
    unsigned int a[4];
    unsigned int precalc_a = structX0[blockIdx.x].precalc[threadIdx.x / 8];
    unsigned long X0 = structX0[blockIdx.x].X0;
    for (int k = 0; k < 4; k++) {
        a[k] = ai_device[k][threadIdx.x] | ((precalc_a >> (4 * (threadIdx.x % 8) + k)) & 1u) << 31;
    }
    int m[4][N];
    for (int k = 0; k < 4; k++) {
        for (int l = 0; l < N; l++) {
            m[k][l] = mi_device[k][l][threadIdx.x];
        }
    }
    bool Ax[4];
    bool stop;
    for (unsigned int X1: X1_device) {
        stop = false;
        for (int i = 0; i < NBR_RIDGES; i += BLOCK_SIZE) r[i + threadIdx.x] = 0;
        __syncthreads();
        for (int j = 0; j < 4; j++) {
            Ax[j] = __popc(a[j] & X1) & 1;
        }
        int count = 0;
        for (bool j: Ax) {
            count += __syncthreads_count(j);
        }
        if (count > MAX_NBR_FACETS) continue;
        for (int j = 0; j < 4; j++) {
            if (stop) continue;
            if (Ax[j]) {
                for (int k = 0; k < N; k++) {
                    if (atomicAdd(r + m[j][k], 1) >= 4) {
                        stop = true;
                        continue;
                    }
                }
            }
        }
        if (__syncthreads_or(stop)) continue;
        if (threadIdx.x == 0) {
            out_device[atomicAdd(&n_out_device, 1)] = (X0 | (unsigned long) (X1^(1u<<31)));
        }
    }

}

void increment_vect(unsigned int vect[], const unsigned int ref[], const int starting_index, const int size) {
    vect[0] = (vect[0] + 1) % ref[starting_index];
    int k = 0;
    while (vect[k] == 0 and k < size - 1) {
        k += 1;
        vect[k] = (vect[k] + 1) % ref[starting_index + k];
    }
}


int main() {
    unsigned int vectX0[sizeVectX0];
    for (int k = 0; k < sizeVectX0; k++) vectX0[k] = 0;
    unsigned int vectX1[sizeVectX1];
    for (int k = 0; k < sizeVectX1; k++) vectX1[k] = 0;
    unsigned int list_shifts[NBR_GROUPS];
    unsigned int list_ref[NBR_GROUPS-1];
    unsigned long list_elementary[NBR_GROUPS-1][N] = {};
    unsigned long X0;

    //Initialiser les matrices ai et mi
    for (int k = 0; k < NBR_FACETS; k++) {
        ai_host[k % 4][k / 4] = (unsigned int) ((A[k] << 32) >> 32);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(ai_device), ai_host, sizeof(ai_host));
    for (int k = 0; k < NBR_FACETS; k++) {
        for (int l = 0; l < N; l++) {
            mi_host[k % 4][l][k / 4] = M[l][k];
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(mi_device), mi_host, sizeof(mi_host));
    //Initialiser les shifts et les générateurs de combinaison linéaire
    list_shifts[NBR_GROUPS-1] = 0;
    for (int k = NBR_GROUPS-2; k > -1; k--) {
        list_shifts[k] = list_groups[k + 1] + list_shifts[k + 1];
    }
    for (int i = 0; i < NBR_GROUPS-1; i++) {
        int position = 0;
        for (int j = 0; j < (1ul << (list_groups[i + 1])); j++) {
            if (__popcount(j) <= 2) {
                unsigned long jl = j;
                list_elementary[i][position] = (jl << list_shifts[i + 1]);
                position += 1;
            }
        }
        list_ref[i] = position;
    }
    //Initialiser les X1
    unsigned int X1;
    for (unsigned int &X1_val: X1_host) {
        X1 = 1u << 31;
        for (int i = 0; i < sizeVectX1; i++) {
            X1 |= (unsigned int) (list_elementary[i + sizeVectX0][vectX1[i]]);
        }
        X1_val = X1;
        increment_vect(vectX1, list_ref, sizeVectX0, sizeVectX1);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(X1_device), X1_host, sizeof(X1_host));
    bool last_one_copied;
    bool first_appeared;
    for (int l = 0; l < NBR_LOOPS; l++) {
        last_one_copied=false;
        for (int index = 0; index < nbrX0; index++) {
            X0 = (1ul << 63);
            for (int i = 0; i < sizeVectX0; i++) {
                X0 |= list_elementary[i][vectX0[i]];
            }
            increment_vect(vectX0, list_ref, 0, sizeVectX0);
        }
        for (int index = 0; index < nbrX0; index++) {
            for (int i = 0; i < BLOCK_SIZE; i++) {
                for (int k = 0; k < 4; k++) {
                    host_listX0[index].precalc[i / 8] |=
                            ((__popcount(host_listX0[index].X0 & A[i * 4 + k])) & 1u) << (4 * (i % 8) + k);
                }
            }
        }
        hipMemcpyToSymbol(HIP_SYMBOL(device_listX0), host_listX0, sizeof(host_listX0));
        kernel<<<1, BLOCK_SIZE>>>(device_listX0);
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
            printf("kernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
        hipMemcpyFromSymbol(&n_out_host, HIP_SYMBOL(n_out_device), sizeof(n_out_device));
        if (n_out_host > (1u << 24)) {
            hipMemcpyFromSymbol(&n_out_host, HIP_SYMBOL(n_out_device), sizeof(n_out_device));
            hipMemcpyFromSymbol(&out_host, HIP_SYMBOL(out_device), sizeof(out_device));
            for (int i = 0; i < n_out_host; i++) {
                first_appeared = false;
                cout<<'[';
                for (int j=0;j<NBR_FACETS;j++){
                    if (__popcount(out_host[i]&A[j])&1ul){
                        if(first_appeared) cout<<',';
                        first_appeared = true;
                        cout<<F[j];
                    }
                }
                cout<<']'<<'\n';
            }
            n_out_host = 0;
            last_one_copied=true;
            hipMemcpyToSymbol(HIP_SYMBOL(n_out_device), &n_out_host, sizeof(n_out_host));
        }
    }
    hipMemcpyFromSymbol(&n_out_host, HIP_SYMBOL(n_out_device), sizeof(n_out_device));
    hipMemcpyFromSymbol(&out_host, HIP_SYMBOL(out_device), sizeof(out_device));
    if (not last_one_copied){
        for (int i = 0; i < n_out_host; i++) {
            cout<<'[';
            first_appeared=false;
            for (int j=0;j<NBR_FACETS;j++){
                if (__popcount(out_host[i]&A[j])&1ul){
                    if (first_appeared) cout<<',';
                    first_appeared = true;
                    cout<<F[j];
                }
            }
            cout<<']'<<'\n';
        }
    }
}